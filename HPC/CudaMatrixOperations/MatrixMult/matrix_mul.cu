#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
 #include <stdio.h>
 #include<hip/hip_runtime.h>
 #include <stdlib.h>
 #include<time.h>
 
 #define SIZE 3
 
 __global__ void matrixmult(int *mat1, int *mat2, int *res)
 {
	 int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	 int COL = blockIdx.x*blockDim.x + threadIdx.x;
	 int mult = 0;
	 for (int i = 0; i < SIZE; i++) {
		 mult += mat1[ROW * SIZE + i] * mat2[i * SIZE + COL];
	 }
	 res[ROW * SIZE + COL] = mult;
 }
 
 int main(void)
 {
	 int i, j;
	 srand(time(NULL));
 //	int a[SIZE][SIZE], b[SIZE][SIZE], c[SIZE][SIZE];
	 int c[SIZE][SIZE];
	 int a[][SIZE]={{1,2,3},{4,5,6},{7,8,9}};
	 int b[][SIZE]={{1,2,3},{4,5,6},{7,8,9}};
 
	 int *dev_a, *dev_b, *dev_c;
 
	 hipMalloc((void **)&dev_a, SIZE*SIZE * sizeof(int));
	 hipMalloc((void **)&dev_b, SIZE*SIZE * sizeof(int));
	 hipMalloc((void **)&dev_c, SIZE*SIZE * sizeof(int));
 
	 printf("\nThe 1st matrix is:\n");
	 for (i = 0; i < SIZE; i++){
		 for (j = 0; j < SIZE; j++){
			 printf("%d\t", a[i][j]);
		 }
		 printf("\n");
	 }
 
	 printf("\nThe 2nd matrix is:\n");
	 for (i = 0; i < SIZE; i++){
		 for (j = 0; j < SIZE; j++){
			 printf("%d\t", b[i][j]);
		 }
		 printf("\n");
	 }
 
	 hipMemcpy(dev_a, a, sizeof(a), hipMemcpyHostToDevice);
	 hipMemcpy(dev_b, b, sizeof(b), hipMemcpyHostToDevice);
	 matrixmult << <SIZE, SIZE >> > (dev_a, dev_b, dev_c);
	 hipMemcpy(&c, dev_c, sizeof(c), hipMemcpyDeviceToHost);
 
	 printf("\nResult matrix is:\n");
	 for (i = 0; i < SIZE; i++){
		 for (j = 0; j < SIZE; j++){
			 printf("%d\t", c[i][j]);
		 }
		 printf("\n");
	 }
 
	 return 0;
 }
 