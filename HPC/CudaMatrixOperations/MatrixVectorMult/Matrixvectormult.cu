/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
 #include <stdio.h>
 #include<hip/hip_runtime.h>
 #include <stdlib.h>
 #include<time.h>
 
 #define SIZE 3
 
 __global__ void matrixvectmult(int *mat,int *vect,int *res)
 {
	 int tid=blockIdx.x*blockDim.x;
	 int mult=0;
	 for(int i=0;i<SIZE;i++)
	 {
		 mult=mult+(mat[tid+i]*vect[i]);
	 }
	 res[blockIdx.x]=mult;
 }
 
 int main(void)
 {
	 int i,j;
	 srand(time(NULL));
	 //int a[SIZE][SIZE],b[SIZE],c[SIZE];
	 int c[SIZE];
	 int a[][SIZE]={{1,2,3},{4,5,6},{7,8,9}};
	 int b[]={1,2,3};
 
	 int *dev_a,*dev_b,*dev_c;
 
	 hipMalloc(&dev_a, SIZE*SIZE*sizeof(int));
	 hipMalloc(&dev_b, SIZE*sizeof(int));
	 hipMalloc(&dev_c, SIZE*sizeof(int));
 
	 printf("\nThe matrix is:\n");
	 for(i=0;i<SIZE;i++){
		 for(j=0;j<SIZE;j++){
			 printf("%d\t",a[i][j]);
		 }
		 printf("\n");
	 }
 
	 printf("\nThe vector is:\n");
	 for(i=0;i<SIZE;i++)
		 printf("%d  ",b[i]);
 
	 hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
	 hipMemcpy(dev_b,b,sizeof(b),hipMemcpyHostToDevice);
	 matrixvectmult<<<SIZE,SIZE>>>(dev_a,dev_b,dev_c);
	 hipMemcpy(&c,dev_c,sizeof(c),hipMemcpyDeviceToHost);//&c and c is same
 
	 printf("\nThe result is:\n");
	 for(int i=0;i<SIZE;i++)
		 printf("%d ",c[i]);
 
	 return 0;
 }
 