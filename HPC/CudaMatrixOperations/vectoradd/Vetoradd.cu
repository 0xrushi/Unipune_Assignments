/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
 #include <stdio.h>
 #include<hip/hip_runtime.h>
 #include <stdlib.h>
 #include<time.h>
 
 #define SIZE 10
 
 __global__ void vectsum(int *x,int *y,int *z)
 {
	 int tid=blockIdx.x*blockDim.x+threadIdx.x;
	 z[tid]=x[tid]+y[tid];
 }
 
 int main(void)
 {
	 int i;
	 srand(time(NULL));
 
	 int a[]={2,33,44,1,32,64,23,89,90,67};
	 int b[]={43,67,35,12,22,74,79,89,92,33};
	 int c[SIZE];
 
	 int *dev_a,*dev_b,*dev_c;
 
	 hipMalloc((void **)&dev_a, SIZE*sizeof(int));
	 hipMalloc((void **)&dev_b, SIZE*sizeof(int));
	 hipMalloc((void **)&dev_c, SIZE*sizeof(int));
 
	 printf("\nThe 1st vector is:\n");
	 for(i=0;i<SIZE;i++)
		 printf("%d  ",a[i]);
 
	 printf("\nThe 2nd vector is:\n");
	 for(i=0;i<SIZE;i++)
		 printf("%d  ",b[i]);
 
	 hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
	 hipMemcpy(dev_b,b,sizeof(b),hipMemcpyHostToDevice);
	 vectsum<<<1,SIZE>>>(dev_a,dev_b,dev_c);
	 hipMemcpy(&c,dev_c,sizeof(c),hipMemcpyDeviceToHost);//&c =c as its an array
 
	 printf("\nThe result is:\n");
	 for(int i=0;i<SIZE;i++)
		 printf("%d  ",c[i]);
 
	 return 0;
 }
 