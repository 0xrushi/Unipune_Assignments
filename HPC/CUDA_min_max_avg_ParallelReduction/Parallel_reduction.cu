#include<hip/hip_runtime.h>
#include <stdio.h>
#include<numeric>

#define SIZE 16

__global__ void para_max(int *input)
{
    int tid=threadIdx.x;
    int step_size=1;
    int no_of_thread=blockDim.x;
    while(no_of_thread>0)
    {
        if(tid<no_of_thread)
        {
            int fst=tid*step_size*2;
            int snd=fst+step_size;
            if(input[fst]<input[snd])
              input[fst]=input[snd];
        }
            step_size <<=1;
            no_of_thread >>=1;
    }
}

__global__ void para_min(int *input)
{
    int tid=threadIdx.x;
    int step_size=1;
    int no_of_thread=blockDim.x;
    while(no_of_thread>0)
    {
        if(tid<no_of_thread)
        {
            int fst=tid*step_size*2;
            int snd=fst+step_size;
            if(input[fst]>input[snd])
              input[fst]=input[snd];
        }
            step_size <<=1;
            no_of_thread >>=1;
    }
}

__global__ void para_add(int *input)
{
    int tid=threadIdx.x;
    int step_size=1;
    int no_of_thread=blockDim.x;
    while(no_of_thread>0)
    {
        if(tid<no_of_thread)
        {
            int fst=tid*step_size*2;
            int snd=fst+step_size;
            input[fst]+=input[snd];
        }
            step_size <<=1;
            no_of_thread >>=1;
    }
}

__global__ void para_avg(int *input)
{
    int tid=threadIdx.x;
    int step_size=1;
    int no_of_thread=blockDim.x;
    while(no_of_thread>0)
    {
        if(tid<no_of_thread)
        {
            int fst=tid*step_size*2;
            int snd=fst+step_size;
            input[fst]+=input[snd];
        }
            step_size <<=1;
            no_of_thread >>=1;
    }
	input[0]=input[0]/SIZE;
}
  int main(void)
  {
      int i;
      int result;
      int *dev_a;
      hipMalloc(&dev_a, SIZE*sizeof(int));

      int a[]={1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16};

	  printf("Vector is: ");
      for(int i=0;i<SIZE;i++)
          printf("%d ",a[i]);

      hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
      para_max<<<1, SIZE/2>>>(dev_a);
      hipMemcpy(&result,dev_a,sizeof(result),hipMemcpyDeviceToHost);
      printf("\n Max is: ");
      printf("%d\n",result);

      hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
      para_min<<<1, SIZE/2>>>(dev_a);
      hipMemcpy(&result,dev_a,sizeof(result),hipMemcpyDeviceToHost);
      printf(" Min is: ");
      printf("%d\n",result);

      hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
      para_add<<<1, SIZE/2>>>(dev_a);
      hipMemcpy(&result,dev_a,sizeof(result),hipMemcpyDeviceToHost);
      printf(" Sum is: ");
      printf("%d\n",result);

      hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
      para_avg<<<1, SIZE/2>>>(dev_a);
      hipMemcpy(&result,dev_a,sizeof(result),hipMemcpyDeviceToHost);
      printf(" Avg is: ");
      printf("%d\n",result);

      hipFree(dev_a);

      return 0;

  }
