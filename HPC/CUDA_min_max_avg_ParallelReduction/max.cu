#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define SIZE 10

__global__ void max(int *a , int *c)	// kernel function definition
{
	int i = threadIdx.x;	// initialize i to thread ID

	*c = a[0];
	//printf("a[i] is %d \n",a[i]);
	atomicMax(c,a[i]);
	//printf("max is %d \n",*c);
}

int main()
{
	int i;
	srand(time(NULL));  //makes use of the computer's internal clock to control the choice of the seed

	int a[10]={2,41,21,74,86,45,92,35,49,50};
	int c;

	int *dev_a, *dev_c;   //GPU / device parameters

	hipMalloc((void **) &dev_a, SIZE*sizeof(int));      //assign memory to parameters on GPU
	hipMalloc((void **) &dev_c, SIZE*sizeof(int));

	for( i = 0 ; i < SIZE ; i++)
	{
		a[i] = i; // rand()% 1000 + 1;      // input the numbers
		//printf("%d ",a[i]);
	}

	hipMemcpy(dev_c, &c, sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_a, a, SIZE*sizeof(int),hipMemcpyHostToDevice);  //copy the array from CPU to GPU
	max<<<1,SIZE>>>(dev_a,dev_c);									// call kernel function <<<number of blocks, number of threads
	hipMemcpy(&c, dev_c, sizeof(int),hipMemcpyDeviceToHost);		// copy the result back from GPU to CPU

	printf("\nmax =  %d ",c);

	hipFree(dev_a);		// Free the allocated memory
	hipFree(dev_c);
	printf("");

	return 0;
}
